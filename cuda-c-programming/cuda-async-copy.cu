
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

constexpr size_t kSize = 1000000;

class Stopwatch {
public:
  using TimePoint = decltype(std::chrono::high_resolution_clock::now());
  Stopwatch(): start(std::chrono::high_resolution_clock::now()) {}
  ~Stopwatch() {
    end = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::microseconds>(end - start).count()
              << " us\n";
  }
private:
  TimePoint start;
  TimePoint end;
};


void Transfer0(float* orig, float* target0, float* target1, float* target2) {
  hipMemcpy(target0, orig, kSize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(target1, orig, kSize * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(target2, orig, kSize * sizeof(float), hipMemcpyHostToDevice);
};

void Transfer1(float* orig, float* target0, float* target1, float* target2) {
  hipStream_t stream[3];
  float* targets[3] = {target0, target1, target2};
  for (int i = 0; i < 3; ++i) {
    hipStreamCreate(&stream[i]);
  }

  for (int i = 0; i < 3; ++i) {
    hipMemcpyAsync(targets[i], orig, kSize * sizeof(float), hipMemcpyHostToDevice, stream[i]);
  }

  for (int i = 0; i < 3; ++i) {
    hipStreamDestroy(stream[i]);
  }
};

int main() {
  float* original = new float[kSize];
  float* target0, *target1, *target2;

  hipMalloc(&target0, sizeof(float) * kSize);
  hipMalloc(&target1, sizeof(float) * kSize);
  hipMalloc(&target2, sizeof(float) * kSize);

  {
    Stopwatch s;
    Transfer0(original, target0, target1, target2);
  }

  {
    Stopwatch s;
    Transfer1(original, target0, target1, target2);
  }

  hipFree(target0);
  hipFree(target1);
  hipFree(target2);
  
  return 0;
}